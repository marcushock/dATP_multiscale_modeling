#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------------//
//                    |                                       |                         //
//                    |          Function Name                |                         //
//                    |          force_pCa_curve ()           |                         //
//                    |                                       |                         //
//--------------------------------------------------------------------------------------//
#include "force_pCa_curve.h"
#include "problemDefines.h"
//#include "experimentalDataHost.h"
#include "gpuErrchk.h"
#include <math.h>
#include <stdio.h>
#include <boost/thread.hpp>
#include <boost/atomic.hpp>
//------------------
// Functions used
//-----------------
#include "rates_trans_matrix.h"
#include "update_RUs.h"
#include "repeat_simul.h"
#include "setGPU.h"
//-------------------

static boost::atomic<int> totalThreadsFinishedMallocing(0);
static boost::mutex lock;

//------------------------------------------
// The force_pCa_Curve function definition:
//------------------------------------------

void force_pCa_curve(initParticleArgs & args,
                     unsigned long randSeed,
                     float * ForceArrays,
                     float * Fss,
                     float * McArrays,
                     float * CArrays,
                     float * BArrays,
                     float * SRArrays,
                     int cc
                    )
{

// grab a new GPU to balance load
int GPUid = getGPU();
setGPU(GPUid);
// select beginning of this loop's Force array
float * Force = &(ForceArrays[cc * MAX_TSTEPS]);
float * Mc = &(McArrays[cc * MAX_TSTEPS]);
float * C = &(CArrays[cc * MAX_TSTEPS]);
float * B  = &(BArrays[cc * MAX_TSTEPS]);
float * SR  = &(SRArrays[cc * MAX_TSTEPS]);
float * kB_plus;
gpuErrchk(hipMallocManaged(&kB_plus, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(kB_plus, 0, sizeof(float)*N_S*N_S));
float * kB_minus;
gpuErrchk(hipMallocManaged(&kB_minus, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(kB_minus, 0, sizeof(float)*N_S*N_S));
float * k2_plus_dATP;
gpuErrchk(hipMallocManaged(&k2_plus_dATP, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(k2_plus_dATP, 0, sizeof(float)*N_S*N_S));
float * k2_plus_ATP;
gpuErrchk(hipMallocManaged(&k2_plus_ATP, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(k2_plus_ATP, 0, sizeof(float)*N_S*N_S));
float * k2_minus;
gpuErrchk(hipMallocManaged(&k2_minus, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(k2_minus, 0, sizeof(float)*N_S*N_S));
float * k4_plus_dATP;
gpuErrchk(hipMallocManaged(&k4_plus_dATP, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(k4_plus_dATP, 0, sizeof(float)*N_S*N_S));
float * k4_plus_ATP;
gpuErrchk(hipMallocManaged(&k4_plus_ATP, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(k4_plus_ATP, 0, sizeof(float)*N_S*N_S));
float * k4_minus;
gpuErrchk(hipMallocManaged(&k4_minus, sizeof(float)*N_S*N_S));
gpuErrchk(hipMemset(k4_minus, 0, sizeof(float)*N_S*N_S));

float gamma_B = args.gamma_B;
float gamma_M = args.gamma_M;
float mu_B = args.gamma_M;
float mu_M = args.mu_M;
float kB_plus_ref = args.kB_plus_ref;
float kB_minus_ref = args.kB_minus_ref;
float k2_plus_ref_dATP = args.k2_plus_ref;
float k3_plus_dATP   = args.k3_plus;
float k4_plus_ref_dATP = args.k4_plus_ref;
float percent_dATP = args.percent_dATP;
float kCa_plus_ref = args.kCa_plus_ref;
float kCa_minus_ref = args.kCa_minus_ref;
float k_force_dATP = args.k_force;
float k_plus_SR_ref_dATP = args.k_plus_SR_ref;
float k_minus_SR_ref = args.k_minus_SR_ref;
int protocol = args.protocol;

float k2_plus_ref_ATP = 0.0025; // parameter defined here
float k3_plus_ATP = 0.05; // parameter defined here
float k4_plus_ref_ATP = 0.135; // parameter defined here
float k_plus_SR_ref_ATP = 16; // parameter defined here
float k_force_ATP = 0.2; // parameter defined here

//-------------------------------
//   Set rates using the input arguments
//-------------------------------
float r = 1; // parameter defined here
float q = 1; // parameter defined here
// float lambda = 0;
float lambda = args.lambda; 
// calculating rates for XB cycling - use Tanner 2007/ Daniel 1998/ Pate & Cooke 1989
float k2_minus_ref, k3_minus, k4_minus_ref;
float conc_ADP,conc_Pi, conc_ATP, x_preR, g_Ca, g_Cb, g_Mc, g_Md, delta_G_ATP, delta_G, k_xb, x_xb;
//float  A, B, C, D, M, N, P, x_b0;
//metabolite concentrations in cytosol
conc_ADP    = 30;        //uM, Dawson et al 1978/ Kushmerick et al 1969 (frog)
conc_ATP    = 3e3;        //uM
conc_Pi     = 3e3;         //uM
// parameter defined here
//thermodynamic parameters
//r_gas         = 8.314;      // Gas constant, J/mol*K
//tc            = 15;
//temp          = tc + 273;   //temperature in Kelvin


// other constants
float alpha = 0.28; // parameter defined here
float eta = 0.68; // parameter defined here
//A = 2000; 
//B = 100; // all from Tanner et al, 2007.
//C = 1;
//D = 1;
//M = 3600;
//N = 40;
//P = 20;
k_xb = 5; // parameter defined here

delta_G_ATP = 13; // units = RT
delta_G = delta_G_ATP - log(conc_ATP/(conc_ADP*conc_Pi)); // units = RT

x_preR      = 0; // XB distortion when pre-rotated.
x_xb        = 0.075;        // nm, XB distortion
//x_b0        = eta * delta_G / k_xb; // xb distortion due to ATP hydrolysis



g_Cb    =  0                                    ;//free energy of XB state Cb
g_Mc    = alpha * delta_G + k_xb * (x_preR)     ;//free energy of XB state Mc
g_Md    = eta* delta_G + k_xb*pow(x_xb,2)       ;//free energy of XB state Md
g_Ca    =   g_Cb                                ;//free energy of XB state Ca


// to get reverse values, keep in mind that rij/rji = e^(gi - gj)
// so, to find r21 = r12/e^(g1 - g2)

//kCa_plus_ref    = 0.09;
//kCa_minus_ref   = 0.113;                    //X_kCa_minus_ref_PSO[i];
//kB_minus_ref    = 0.327;                    //X_kB_minus_ref_PSO[i];
//k2_plus_ref     = A * pow(k_xb/2*M_PI,0.5)*exp(-k_xb*pow(x_preR-x_b0,2)/2); // from tanner 2007
k2_minus_ref    = k2_plus_ref_ATP/ exp(g_Cb - g_Mc);//0.5 / exp(g_Cb - g_Mc);    //using vals from optimization_0227 (k2_plus = 0.615440)
//k3_plus         = (B/pow(k_xb,.5))*(1-tanh(C*pow(k_xb,.5)*(x_xb-x_b0)))+D;        //X_k3_plus_PSO[i];
k3_minus        = k3_plus_ATP / exp(g_Mc - g_Md) ;//0.3 / exp(g_Mc - g_Md);  //
//k4_plus_ref     = pow(k_xb,0.5)*(pow(M*pow(x_xb,2),0.5)-N*x_xb)+ P;                 //X_k4_plus_PSO[i];
k4_minus_ref    = k4_plus_ref_ATP * exp(g_Ca - g_Md - delta_G);

//-------------------------------------
// Call the transition rates function:
//-------------------------------------

rates_trans_matrix(N_S,
kB_plus_ref,
kB_minus_ref,
k2_plus_ref_dATP,
k2_plus_ref_ATP,
k2_minus_ref,
k4_plus_ref_dATP,
k4_plus_ref_ATP,
k4_minus_ref,
gamma_B,
gamma_M,
mu_B,
mu_M,
r,
q,
kB_plus,
kB_minus,
k2_plus_dATP,
k2_plus_ATP,
k2_minus,
k4_plus_dATP,
k4_plus_ATP,
k4_minus
);

    //-----------------------------------------------------
    // start Ca- loop i.e., to get the entire F-Ca curve:
    //-----------------------------------------------------

    float Ftemp        = 0.0;                      // is used to calculate the steady-state force at the end
    float Calc_conc_exp           = pow(10.0f,-(args.experimentalData[cc].first-6));     // Ca2+ concentration in uM
    // float kCa_plus     = Cal_conc*kCa_plus_ref;
    float kCa_plus     = kCa_plus_ref;
    float kCa_minus    = kCa_minus_ref;
    const int n_pCa = args.experimentalData.size();
    
    
    //---------------------------------
    // Call the repeat_simul function:
    //----------------------------------
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    totalThreadsFinishedMallocing++;
    while(totalThreadsFinishedMallocing < n_pCa){
        boost::thread::yield();
    }
repeat_simul<<<MAX_REPS/32, 32, 0, s>>>(lambda,
                                        randSeed,
                                        k4_plus_dATP,
                                        k4_plus_ATP,
                                        k4_minus,
                                        k3_plus_dATP,
                                        k3_plus_ATP,
                                        k3_minus,
                                        k2_plus_dATP,
                                        k2_plus_ATP,
                                        k2_minus,
                                        kB_plus,
                                        kB_minus,
                                        kCa_plus_ref,
                                        kCa_minus_ref,
                                        percent_dATP,
                                        k_force_dATP,
                                        k_force_ATP,
					k_plus_SR_ref_dATP,
					k_plus_SR_ref_ATP,
					k_minus_SR_ref,
                                        Force,
                                        Mc,
                                        C,
                                        B,
                                        SR,
                                        cc, 
                                        protocol, 
                                        Calc_conc_exp
                                           );

    gpuErrchk(hipStreamSynchronize(s)); // wait for device to finish repeat_simul
    gpuErrchk(hipStreamDestroy(s));
    //--------------------------------------------------------------------------------------
    // Calculate The Steady-State Force using Impluse using data from the last 5 sec (was previously 0.5)
    // (i.e., just 100000 time steps) only using numerical trapaziodal integration
    //--------------------------------------------------------------------------------------

    for (int n = MAX_TSTEPS-1000000; n < MAX_TSTEPS-1; n++)  // time marching Originally was set to 100000
    {
        Ftemp = Ftemp+Force[n];
    }

    Fss[cc] = (Ftemp + (0.5f * Force[MAX_TSTEPS-1000001]) + (0.5f * Force[MAX_TSTEPS-1])) / 1000000.0f / MAX_REPS;    //Fss[cc] = 1;

    //--------------------------------

    // free allocated memory
gpuErrchk(hipFree(kB_plus));
gpuErrchk(hipFree(kB_minus));
gpuErrchk(hipFree(k2_plus_dATP));
gpuErrchk(hipFree(k2_plus_ATP));
gpuErrchk(hipFree(k2_minus));
gpuErrchk(hipFree(k4_plus_dATP));
gpuErrchk(hipFree(k4_plus_ATP));
gpuErrchk(hipFree(k4_minus));

    lock.lock();
    if(totalThreadsFinishedMallocing == n_pCa){
        totalThreadsFinishedMallocing = 0;
    }
    lock.unlock();

} // end main function
