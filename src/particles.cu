#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------------//
//                    |                                       |                         //
//                    |          Function Name                |                         //
//                    |          force_pCa_curve ()           |                         //
//                    |                                       |                         //
//--------------------------------------------------------------------------------------//
#include "particles.h"
#include "problemDefines.h"
#include "setGPU.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <boost/thread.hpp>

#include "force_pCa_curve.h"
//#include "experimentalDataHost.h"
#include "gpuErrchk.h"

inline float getRand()
{
    return (float) (RANDVAL) / (float) (RAND_MAX);
}

void init_particle(initParticleArgs & args)
{
    const int n_pCa = args.experimentalData.size();


    //-----------------------------------------------------------------------------------------
    // Call the force_pCa_curve function to get the force as a function of Ca++ concentrations.
    // NB: this function implicitly calls the other functions.
    //-----------------------------------------------------------------------------------------

    // allocate arrays for Force outside of kernel to use global GPU memory
    float * ForceArrays_return;
    gpuErrchk(hipMallocManaged(&ForceArrays_return, sizeof(float)*n_pCa*MAX_TSTEPS));
    gpuErrchk(hipMemset(ForceArrays_return, 0, sizeof(float)*n_pCa*MAX_TSTEPS));
    float * Fss_return;
    gpuErrchk(hipMallocManaged(&Fss_return, sizeof(float)*n_pCa));
    gpuErrchk(hipMemset(Fss_return, 0, sizeof(float)*n_pCa));
    float * McArrays_return;
    gpuErrchk(hipMallocManaged(&McArrays_return, sizeof(float)*n_pCa*MAX_TSTEPS));
    gpuErrchk(hipMemset(McArrays_return, 0, sizeof(float)*n_pCa*MAX_TSTEPS));
    float * CArrays_return;
    gpuErrchk(hipMallocManaged(&CArrays_return, sizeof(float)*n_pCa*MAX_TSTEPS));
    gpuErrchk(hipMemset(CArrays_return, 0, sizeof(float)*n_pCa*MAX_TSTEPS));
    float * BArrays_return;
    gpuErrchk(hipMallocManaged(&BArrays_return, sizeof(float)*n_pCa*MAX_TSTEPS));
    gpuErrchk(hipMemset(BArrays_return, 0, sizeof(float)*n_pCa*MAX_TSTEPS));
    float * SRArrays_return;
    gpuErrchk(hipMallocManaged(&SRArrays_return, sizeof(float)*n_pCa*MAX_TSTEPS));
    gpuErrchk(hipMemset(SRArrays_return, 0, sizeof(float)*n_pCa*MAX_TSTEPS));

    std::cout << "Running force_pCa_curve" << std::endl;
    boost::thread_group pcaThreadGroup;
    for(int cc = 0; cc < n_pCa; ++cc)
    {
        pcaThreadGroup.add_thread(new boost::thread(force_pCa_curve,
            args,
            RANDVAL,
            ForceArrays_return,
            Fss_return,
            McArrays_return,
            CArrays_return,
            BArrays_return,
            SRArrays_return,
            cc));
    }
    pcaThreadGroup.join_all();

    float Fss_max_temp = 0.0;                         // is used to calculate the max s-s force
    //------------------------------------------------------------
    // Obtain the maximum vale of all the steady-state force values
    //-------------------------------------------------------------
    for(int j = 0; j < n_pCa; ++j)
    {
        if (Fss_return[j] > Fss_max_temp)
        {
            Fss_max_temp = Fss_return[j];
        }
    }

    //-------------------------------------
    // Formulate Residual/Cost Function :
    //--------------------------------------
    float residual_temp = 0.0;
    for (int j = 0; j < n_pCa; ++j)  // Ca-loop
    {
        // experimentalData[j].second == F_exp[j]
        residual_temp += pow((args.experimentalData[j].second - Fss_return[j] / Fss_max_temp),2); // normalized force
    }

    // set return residual
    float force_pCa_residual = pow(residual_temp,0.5f);
    std::cout << "Residual: " << force_pCa_residual << std::endl;

        std::string dataAppend =
        //" percent " + std::to_string(args.percent_dATP) +
        //" gammaB " + std::to_string(args.gamma_B) +
        //" gammaM " + std::to_string(args.gamma_M) +
        //" mu_M " + std::to_string(args.mu_M) +
        " k2_plus_ref " + std::to_string(args.k2_plus_ref) +
        " k3_plus " + std::to_string(args.k3_plus) +
        " k4_plus_ref " + std::to_string(args.k4_plus_ref) +
        " kB_plus_ref " + std::to_string(args.kB_plus_ref) +
        " kB_minus_ref " + std::to_string(args.kB_minus_ref) +
        //" lambda " + std::to_string(args.lambda) +
        " kCa_plus_ref " + std::to_string(args.kCa_plus_ref) +
        " dATP " + std::to_string(args.percent_dATP) +
        " k_force " + std::to_string(args.k_force) +
        " k_plus_SR_ref " + std::to_string(args.k_plus_SR_ref) +
        " k_minus_SR_ref " + std::to_string(args.k_minus_SR_ref);
        std::string Force_out_Filename = ("MCMC_simulation_results/Force_out"+dataAppend+".csv");
        std::string States_out_Filename = ("MCMC_simulation_results/States_out"+dataAppend+".csv");
        std::string Force_pCa_out_Filename = ("MCMC_simulation_results/Force_pCa_Optmz"+dataAppend+".csv");
        std::string Force_pCa_normalized_out_Filename = ("MCMC_simulation_results/Force_pCa_Optmz_Normalized"+dataAppend+".csv");

        /* raw force out */
        const int skipFactor = 1000;
        std::ofstream Force_out(Force_out_Filename); //opening an output stream for file *.csv
        for(int j = 0; j < MAX_TSTEPS; j+=skipFactor)
        {
            Force_out << DT*j;
            for (int cc = 0; cc < n_pCa; cc++)  // Ca-loop
            {
                Force_out << "," << ForceArrays_return[cc * MAX_TSTEPS + j]/MAX_REPS;

            }
            Force_out << std::endl;
        }
        Force_out.close();
        std::cout << "data successfully saved into the file name: " << Force_out_Filename << std::endl;



        std::ofstream States_out(States_out_Filename); //opening an output stream for file *.csv
        for(int j = 0; j < MAX_TSTEPS; j+=skipFactor)
        {
            States_out << DT*j;
            for (int cc = 0; cc < n_pCa; cc++)  // Ca-loop
            {
                States_out << "," << ForceArrays_return[cc * MAX_TSTEPS + j]/MAX_REPS << "," << McArrays_return[cc * MAX_TSTEPS + j]/MAX_REPS << "," << CArrays_return[cc * MAX_TSTEPS + j]/MAX_REPS << "," << BArrays_return[cc * MAX_TSTEPS + j]/MAX_REPS << "," << SRArrays_return[cc * MAX_TSTEPS + j]/MAX_REPS;

            }
            States_out << std::endl;
        }
        States_out.close();
        std::cout << "data successfully saved into the file name: " << States_out_Filename << std::endl;





        /* end raw force out */

        /* pca + pca normalized out */
        std::ofstream Force_pCa_out(Force_pCa_out_Filename); //opening an output stream for file *.csv
        std::ofstream Force_pCa_normalized_out(Force_pCa_normalized_out_Filename); //opening an output stream for file *.csv
        for (int cc = 0; cc < n_pCa; cc++)  // Ca-loop
        {
            // experimentalData[cc].first == pCa[cc]
            Force_pCa_out << args.experimentalData[cc].first << "," << Fss_return[cc] << std::endl; // write the average force at each time in a file
            Force_pCa_normalized_out << args.experimentalData[cc].first << "," << Fss_return[cc] / Fss_max_temp << std::endl; // normalized force
        }
        Force_pCa_out.close();
        std::cout << " data successfully saved into the file name: " << Force_pCa_out_Filename << std::endl;
        Force_pCa_normalized_out.close();
        std::cout << " data successfully saved into the file name: " << Force_pCa_normalized_out_Filename << std::endl;
        /* end pca + pca normalized out */

    gpuErrchk(hipFree(ForceArrays_return));
    gpuErrchk(hipFree(Fss_return));
    gpuErrchk(hipFree(McArrays_return));
    gpuErrchk(hipFree(CArrays_return));
    gpuErrchk(hipFree(BArrays_return));
    gpuErrchk(hipFree(SRArrays_return));

}
