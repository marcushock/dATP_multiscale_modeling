#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------------//
//                    |                                       |                         //
//                    |          Function Name                |                         //
//                    |         repeat_simul()                |                         //
//                    |                                       |                         //
//--------------------------------------------------------------------------------------//
//---------------------------------------------------------------------------------
// This function is used to update the states of each RUs based on the Markov step
//---------------------------------------------------------------------------------
// Input|
//-------
// as shown below
//--------
// Output|
//---------
// Force-Ca curve at at a given Ca value
//--------------------------------------------------------------------------%
#include "repeat_simul.h"
#include "problemDefines.h"
#include "lin_interp_ca.h"
//--------------------------------------
// functions used within this function
//--------------------------------------
#include "rates_trans_matrix.h"
#include "update_RUs.h"
#include "genrand.h"
#include <stdio.h>
//-----------------------------------------------
// This function definition
//-----------------------------------------------

__global__ void repeat_simul(float lambda,
const unsigned long randSeed,
float * k4_plus_dATP,
float * k4_plus_ATP,
float * k4_minus,
float k3_plus_dATP,
float k3_plus_ATP,
float k3_minus,
float * k2_plus_dATP,
float * k2_plus_ATP,
float * k2_minus,
float * kB_plus,
float * kB_minus,
float kCa_plus_ref,
float kCa_minus,
float percent_dATP,
float k_force_dATP,
float k_force_ATP,
float k_plus_SR_ref_dATP,
float k_plus_SR_ref_ATP,
float k_minus_SR_ref,
float * Force,
float * Mc,
float * C,
float * B,
float * SR,
int cc
)

{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    /* initialize random number generation per thread */
    float randNum[N_RU];
    float rand_dATP[N_RU];
    int RU[N_RU];
    bool caRU[N_RU];
    hiprandState_t state;
    float f;
    int flag = 0;
    float f_prev;
    float SR_prev;
    float kCa_plus;
    float calcium;
    float current_time;


    hiprand_init(randSeed, index, 0, &state);    
//--------------------------------------
    // start repeat loop i.e., using r-index
    //--------------------------------------

    //reset RUs again to B0
    memset(RU, 0, sizeof(int)*N_RU);
    memset(caRU, 0, sizeof(bool)*N_RU);
    RU[0]=2;
    RU[N_RU-1]=2;
    
    for(int i = 1; i < N_RU-1; ++i)
    {
    	RU[i]=0;
    }
    //------------------------------------
    // start time loop i.e., using n-index
    //------------------------------------
    for (int n = 0; n < MAX_TSTEPS; ++n)  // time marching
    {
        // begin n-loop for time marching
        int count_Md_state  = 0;                // used to find how many Md-state in each iteration
        int count_Mc_state  = 0;
        int count_C_state  = 0;
        int count_B_state   = 0;
        int count_SR_state = 0;
        genrand(randNum, N_RU, &state); // fills array with random numbers
        genrand(rand_dATP, N_RU, &state); // fills array with random numbers
        current_time = n*DT;
        calcium = lin_interp_ca(current_time);
        kCa_plus = kCa_plus_ref * calcium;
        //-----------------------------------
        // call the updated RUs
        //-----------------------------------
        
        // Ktr protocol
	//if (n == 3000001)
	//	{
	//	for (int y = 0; y<N_RU; ++y)
	//		{
	//		RU[y] = 0;
	//		}
	//	}

        
        
        if(n==0)
        {
        	f = 0;
        }
        // NOTE: This has been commented out, because I believe that this was the cause of the max_repeats issue. 
        // Force is eventually normalized when it saved, however, at this point, with all of the repeats 
        // running simultaneously, the Force array is inflated when there are more repeats running. 
        // Instead, we are getting the previous fraction of force states from the filament via the 
        // code below after counting the states (f = forceValue;)


        // else
        // {
        // 	f = (float)Force[n-1];
        // }
        // float current_max=0.0;
        // if (current_max < f){
        //     current_max = f;
        //     printf("New_max = %f, %i\n",f, cc);
        // }
        
        float k_plus_SR_ATP = k_plus_SR_ref_ATP; //*(1+k_force_ATP*f);
        float k_plus_SR_dATP = k_plus_SR_ref_dATP; //*(1+k_force_dATP*f);
        float k_minus_SR = k_minus_SR_ref;
        //printf("%f\n",k_plus_SR);
        //printf("%f\n",k_minus_SR);

       update_RUs(lambda, DT, kCa_plus, kCa_minus, randNum, rand_dATP, RU, caRU, kB_plus, kB_minus, k2_plus_dATP, k2_plus_ATP, k2_minus, k3_plus_dATP, k3_plus_ATP, k3_minus, k4_plus_dATP, k4_plus_ATP, k4_minus, percent_dATP, k_force_dATP, k_force_ATP, k_plus_SR_dATP, k_plus_SR_ATP, k_minus_SR,f);

        //--------------------------------------------
        // Obtain Force estimate based on the M-state
        //--------------------------------------------
        for(int i = 0; i < N_RU; ++i)
        {
            if(RU[i]==5) // this represents M2
            {
                ++count_Md_state;
            }
            else if(RU[i]==4) // this represents M1
            {
                ++count_Mc_state;
            }
            else if(RU[i]==3) // this represents C
            {
                ++count_C_state;
            }
            else if (RU[i]==2) // this represents B
            {
                ++count_B_state;
            }
            else if (RU[i]==1) // this represents C* (SRX)
            {
                ++count_SR_state;
            }
            else if (RU[i]==0) // this represents B* (SRX)
            {
                ++count_SR_state;
            }
        }
        float forceValue = (float)count_Md_state / (N_RU); // Type casting because count_Md_state is defined as an int 
        float McValue = (float)count_Mc_state / (N_RU);
        float CValue = (float)count_C_state / (N_RU);
        float BValue = (float)count_B_state / (N_RU);
        float SRValue = (float)count_SR_state / (N_RU);
        
        f = forceValue;
        // float current_max = 0;
        // // This is to look at what happens after the there is an instance where there is at least one state in the force producing state 
        // // It also looks at the following state to see if everything transitions out. 

        // if (flag == 1){
        //     // % ['Count', 'f_prev','SR_prev','f','SRValue']
        //     printf("%i, %f, %f, %f, %f\n",n, f_prev, SR_prev, f, SRValue);
        //     // printf("current SRValue = %f, %i\n", f, n);
        // }

        // if (f > 0) {
        //     flag = 1;
        //     // printf("current f = %f, %i\n",f, n);
        //     // printf("current SRValue = %f, %i\n", f, n);
        //     f_prev = f;
        //     SR_prev = SRValue;

        // }
        // else {
        //     flag = 0;
        // }

        if ( n % 10000 == 0){
            printf("current ca = %f, %f\n",calcium, current_time);
        }

        


        atomicAdd(&(Force[n]), forceValue); // add results every repeat
        atomicAdd(&(Mc[n]), McValue); // add results every repeat
        atomicAdd(&(C[n]), CValue); // add results every repeat
        atomicAdd(&(B[n]), BValue); // add results every repeat
        atomicAdd(&(SR[n]), SRValue); // add results every repeat
    } // end the (n-loop) of the time marching
}
